#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "bmpfile.h"

// Define constants
#define VERBOSE 1
#define FILENAME "my_mandelbrot_fractal.bmp"

/*Mandelbrot values*/
#define RESOLUTION 8700.0
#define XCENTER -0.55
#define YCENTER 0.6
#define MAX_ITER 1000
#define MIN_WIDTH 100
#define MAX_WIDTH 19200
#define MIN_HEIGHT 100
#define MAX_HEIGHT 10800

// Define success and failure return values
#define SUCCESS 1
#define FAILURE 0

/*Colour Values*/
#define COLOUR_DEPTH 255
#define COLOUR_MAX 240.0
#define GRADIENT_COLOUR_MAX 230.0

// function definitions - see below main() for function body
int parseArgs(int argc, char *argv[], int *width, int *height);

/**
 * Computes the color gradient
 * color: the output vector
 * x: the gradient (between 0 and 360)
 * min and max: variation of the RGB channels (Move3D 0 -> 1)
 * Check wiki for more details on the colour science: en.wikipedia.org/wiki/HSL_and_HSV
 */
void GroundColorMix(double* color, double x, double min, double max) {
	/*
	 * Red = 0
	 * Green = 1
	 * Blue = 2
	 */
	double posSlope = (max - min) / 60;
	double negSlope = (min - max) / 60;

	if (x < 60) {
		color[0] = max;
		color[1] = posSlope * x + min;
		color[2] = min;
		return;
	} else if (x < 120) {
		color[0] = negSlope * x + 2.0 * max + min;
		color[1] = max;
		color[2] = min;
		return;
	} else if (x < 180) {
		color[0] = min;
		color[1] = max;
		color[2] = posSlope * x - 2.0 * max + min;
		return;
	} else if (x < 240) {
		color[0] = min;
		color[1] = negSlope * x + 4.0 * max + min;
		color[2] = max;
		return;
	} else if (x < 300) {
		color[0] = posSlope * x - 4.0 * max + min;
		color[1] = min;
		color[2] = max;
		return;
	} else {
		color[0] = max;
		color[1] = min;
		color[2] = negSlope * x + 6 * max;
		return;
	}
}

/* Mandelbrot Set Image Demonstration
 *
 * This is a simple single-process/single thread implementation
 * that computes a Mandelbrot set and produces a corresponding
 * Bitmap image. The program demonstrates the use of a colour
 * gradient
 *
 * This program uses the algorithm outlined in:
 *   "Building Parallel Programs: SMPs, Clusters And Java", Alan Kaminsky
 *
 * This program requires libbmp for all bitmap operations.
 *
 */

int main(int argc, char **argv) {

	int width, height;

	// parse CL args
	if (!parseArgs(argc, argv, &width, &height)) {
		exit(EXIT_FAILURE);
	}

	fprintf(stderr, "Creating a fractal: %ipx x %ipx\n",width, height);


	bmpfile_t *bmp;
	rgb_pixel_t pixel = { 0, 0, 0, 0 };
	int xoffset = -(width - 1) / 2;
	int yoffset = (height - 1) / 2;
	bmp = bmp_create(width, height, 32);
	int col = 0;
	int row = 0;
	for (col = 0; col < width; col++) {
		for (row = 0; row < height; row++) {

			//Determine where in the mandelbrot set, the pixel is referencing
			double x = XCENTER + (xoffset + col) / RESOLUTION;
			double y = YCENTER + (yoffset - row) / RESOLUTION;

			//Mandelbrot stuff

			double a = 0;
			double b = 0;
			double aold = 0;
			double bold = 0;
			double zmagsqr = 0;
			int iter = 0;
			double x_col;
			double color[3];
			//Check if the x,y coord are part of the mendelbrot set - refer to the algorithm
			while (iter < MAX_ITER && zmagsqr <= 4.0) {
				++iter;
				a = (aold * aold) - (bold * bold) + x;
				b = 2.0 * aold * bold + y;

				zmagsqr = a * a + b * b;

				aold = a;
				bold = b;

			}

			/* Generate the colour of the pixel from the iter value */
			/* You can mess around with the colour settings to use different gradients */
			/* Colour currently maps from royal blue to red */
			x_col =
					(COLOUR_MAX
							- ((((float) iter / ((float) MAX_ITER)
									* GRADIENT_COLOUR_MAX))));
			GroundColorMix(color, x_col, 1, COLOUR_DEPTH);
			pixel.red = color[0];
			pixel.green = color[1];
			pixel.blue = color[2];
			bmp_set_pixel(bmp, col, row, pixel);

		}

	}

	bmp_save(bmp, FILENAME);
	bmp_destroy(bmp);

	fprintf(stderr, "Complete!\nFractal saved to file: ./%s\n",FILENAME);

	return 0;
}

/*
 * Parse and validate command line args
 *
 * Arguments:
 * int argc - number of arguments supplied
 * char *argv[] - array of command line arguments
 * int *width - width of output file
 * int *height - height of output file
 *
 * Return: FAILURE or SUCCESS values defined above
 */
int parseArgs(int argc, char *argv[], int *width, int *height) {
	// validate the number of args
	if (argc == 3) { // parse and validate args

		// parse and validate the image width
		if ((*width = atoi(argv[1])) < MIN_WIDTH || *width > MAX_WIDTH) {
			fprintf(stderr, "Width of image must be between %i and %i\n",
			MIN_WIDTH, MAX_WIDTH);
			return (FAILURE);
		}

		// parse and validate the image height
		if ((*height = atoi(argv[2])) < MIN_HEIGHT || *height > MAX_HEIGHT) {
			fprintf(stderr, "Height of image must be between %i and %i\n",
			MIN_HEIGHT, MAX_HEIGHT);
			return (FAILURE);
		}

	} else { // incorrect number of args supplied
		fprintf(stderr, "Usage: %s image_width image_height\n", argv[0]);
		return (FAILURE);
	}

	return (SUCCESS);
}
